
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_YCbCr422_8u_C2P3R(uchar1* pSrc, int srcStep,
	uchar1* pDst[3], int dstStep[3], int nWidth, int nHeight) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < nWidth && y < nHeight) {
		int nSrcIdx = y * srcStep + x * 4;
		int nDst0Idx = y * dstStep[0] + x * 2;
		int nDst1Idx = y * dstStep[1] + x;
		int nDst2Idx = y * dstStep[2] + x;

		pDst[0][nDst0Idx] = pSrc[nSrcIdx];
		pDst[0][nDst0Idx + 1] = pSrc[nSrcIdx + 2];
		pDst[1][nDst1Idx] = pSrc[nSrcIdx + 1];
		pDst[2][nDst2Idx] = pSrc[nSrcIdx + 3];
	}
}

extern hipError_t zppiYCbCr422_8u_C2P3R(
	uchar1* pSrc, int srcStep, uchar1* pDst[3], int dstStep[3], int nWidth, int nHeight) {
	static int BLOCK_W = 16;
	static int BLOCK_H = 16;

	nWidth /= 2;

	dim3 grid((nWidth + BLOCK_W-1) / BLOCK_W, (nHeight + BLOCK_H-1) / BLOCK_H, 1);
	dim3 block(BLOCK_W, BLOCK_H, 1);

	kernel_YCbCr422_8u_C2P3R<<<grid, block>>>(
		pSrc, srcStep,
		pDst, dstStep,
		nWidth, nHeight);

	return hipDeviceSynchronize();
}
