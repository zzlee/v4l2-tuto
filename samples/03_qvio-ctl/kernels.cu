
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_Copy_8u_C1R(uchar1* pSrc, int srcStep, uchar1* pDst, int dstStep, int nWidth, int nHeight) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < nWidth && y < nHeight) {
		pDst[y * dstStep + x] = pSrc[y * srcStep + x];
	}
}
