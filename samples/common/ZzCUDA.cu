
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_YCbCr422_8u_C2P3R(uchar1* pSrc, int srcStep, uchar1* pDst0, int dst0Step,
	uchar1* pDst1, int dst1Step, uchar1* pDst2, int dst2Step, int nWidth, int nHeight) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < nWidth && y < nHeight) {
		int nSrcIdx = y * srcStep + x * 4;
		int nDst0Idx = y * dst0Step + x * 2;
		int nDst1Idx = y * dst1Step + x;
		int nDst2Idx = y * dst2Step + x;

		// Y1
		pDst0[nDst0Idx + 0] = pSrc[nSrcIdx + 0];
		// Cb
		pDst1[nDst1Idx + 0] = pSrc[nSrcIdx + 1];
		// Y2
		pDst0[nDst0Idx + 1] = pSrc[nSrcIdx + 2];
		// Cr
		pDst2[nDst2Idx + 0] = pSrc[nSrcIdx + 3];
	}
}

extern hipError_t zppiYCbCr422_8u_C2P3R(uchar1* pSrc, int srcStep, uchar1* pDst[3], int dstStep[3], int nWidth, int nHeight) {
	static int BLOCK_W = 16;
	static int BLOCK_H = 16;

	nWidth /= 2;

	dim3 grid((nWidth + BLOCK_W-1) / BLOCK_W, (nHeight + BLOCK_H-1) / BLOCK_H, 1);
	dim3 block(BLOCK_W, BLOCK_H, 1);

	kernel_YCbCr422_8u_C2P3R<<<grid, block>>>(
		pSrc, srcStep,
		pDst[0], dstStep[0],
		pDst[1], dstStep[1],
		pDst[2], dstStep[2],
		nWidth, nHeight);

	return hipDeviceSynchronize();
}

__global__ void kernel_CbCr422_CbCr420_8u_P2C2R(uchar1* pSrc0, int src0Step, uchar1* pSrc1, int src1Step,
	uchar1* pDst, int dstStep, int nWidth, int nHeight) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < nWidth && y < nHeight) {
		int nSrc0Idx = y * src0Step + x;
		int nSrc1Idx = y * src1Step + x;
		int nDstIdx = y * dstStep + x * 2;

		// Cb
		pDst[nDstIdx + 0] = pSrc0[nSrc0Idx];
		// Cr
		pDst[nDstIdx + 1] = pSrc1[nSrc1Idx];
	}
}

extern hipError_t zppiCbCr422_CbCr420_8u_P2C2R(uchar1* pSrc[2], int srcStep[2], uchar1* pDst, int dstStep, int nWidth, int nHeight) {
	static int BLOCK_W = 16;
	static int BLOCK_H = 16;

	dim3 grid((nWidth + BLOCK_W-1) / BLOCK_W, (nHeight + BLOCK_H-1) / BLOCK_H, 1);
	dim3 block(BLOCK_W, BLOCK_H, 1);

	kernel_CbCr422_CbCr420_8u_P2C2R<<<grid, block>>>(
		pSrc[0], srcStep[0],
		pSrc[1], srcStep[1],
		pDst, dstStep,
		nWidth, nHeight);

	return hipDeviceSynchronize();
}
